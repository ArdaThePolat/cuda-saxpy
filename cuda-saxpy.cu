#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// SAXPY operation function
__global__ void saxpy(int n, float a, float* x, float* y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

int main(void)
{
    // Print device's properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("GPU Device Name: %s\n", prop.name);
    printf("Max Threads Per Block: %d\n\n", prop.maxThreadsPerBlock);

    int n;
    float a;

    printf("Enter the size of the arrays: ");
    scanf("%d", &n);

    // Allocate memory
    float* x, * y;
    hipMallocManaged(&x, n * sizeof(float));
    hipMallocManaged(&y, n * sizeof(float));

    printf("Enter the scalar value: ");
    scanf("%f", &a);

    // Initialize the vectors with random numbers
    for (int i = 0; i < n; i++) {
        x[i] = rand() / (float)RAND_MAX;
        y[i] = rand() / (float)RAND_MAX;
    }

    // Print the initialized y vector
    printf("\nInitialized y vector:\n");
    for (int i = 0; i < n; i++) {
        printf("%f ", y[i]);
    }
    printf("\n\n");

    // Print the  initialized x vector
    printf("Initialized x vector:\n");
    for (int i = 0; i < n; i++) {
        printf("%f ", x[i]);
    }
    printf("\n\n");

    // Try to find the optimum block and grid size
    int tempx = floor(sqrt(n));
    saxpy << < tempx,(n+tempx)/tempx >> > (n, a, x, y);
    hipDeviceSynchronize();

    // Print the result
    printf("Final y vector:\n");
    for (int i = 0; i < n; i++) {
        printf("%f ", y[i]);
    }
    printf("\n\n");

    // Free allocated memory
    hipFree(x);
    hipFree(y);

    return 0;
}
